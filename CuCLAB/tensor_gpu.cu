#include "tensor_gpu.cuh"
#include <hip/hip_runtime.h>


tensor_gpu::tensor_gpu(const int p_size) : _size(p_size)
{
	_data = init_data(_size);
}

tensor_gpu::tensor_gpu(const tensor_gpu& p_copy)
{
	_size = p_copy._size;
	_data = init_data(_size);
	hipMemcpy(_data, p_copy._data, sizeof(float) * p_copy._size, hipMemcpyDeviceToDevice);
}

tensor_gpu& tensor_gpu::operator=(const tensor_gpu& p_copy)
{
	if (_size != p_copy._size)
	{
		hipFree(_data);
		_data = init_data(_size);
	}
	hipMemcpy(_data, p_copy._data, sizeof(float) * p_copy._size, hipMemcpyDeviceToDevice);
	_size = p_copy._size;

	return *this;
}

tensor_gpu::~tensor_gpu()
{
	_size = 0;
	hipFree(_data);
}

void tensor_gpu::to_gpu(float* p_cpu_data) const
{
	hipError_t error = hipMemcpy(_data, p_cpu_data, sizeof(float) * _size, hipMemcpyHostToDevice);
}

void tensor_gpu::to_cpu(float* p_cpu_data) const
{
	hipError_t error = hipMemcpy(p_cpu_data, _data, sizeof(float) * _size, hipMemcpyDeviceToHost);
}

float* tensor_gpu::init_data(int& p_size)
{
	float* result;
	hipError_t error = hipMalloc(reinterpret_cast<void**>(&result), sizeof(float) * p_size);
	return result;
}