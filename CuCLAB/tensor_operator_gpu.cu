#include "hip/hip_runtime.h"
#include "tensor_operator_gpu.cuh"
#include "cuda_kernels.cuh"
#include <cmath>
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include <hipblas.h>

void tensor_operator_gpu::add(float* p_x, float* p_y, float* p_z, const int p_size)
{
	call_kernel(add_kernel, p_x, p_y, p_z, p_size);
}

void tensor_operator_gpu::const_add(float* p_x, const float p_y, float* p_z, const int p_size)
{
	call_kernel(const_add_kernel, p_x, p_y, p_z, p_size);
}

void tensor_operator_gpu::sub(float* p_x, float* p_y, float* p_z, const int p_size)
{
	call_kernel(sub_kernel, p_x, p_y, p_z, p_size);
}

void tensor_operator_gpu::const_sub(float* p_x, const float p_y, float* p_z, const int p_size)
{
	call_kernel(const_sub_kernel, p_x, p_y, p_z, p_size);
}

void tensor_operator_gpu::const_sub(const float p_x, float* p_y, float* p_z, const int p_size)
{
	call_kernel(const_sub_kernel, p_x, p_y, p_z, p_size);
}

void tensor_operator_gpu::mul(float* p_x, bool p_transpose_x, float* p_y, bool p_transpose_y, float* p_z, int p_rows, int p_common, int p_cols)
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	const hipblasOperation_t transpose_x = p_transpose_x ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	const hipblasOperation_t transpose_y = p_transpose_y ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	const int lda = p_transpose_x ? p_rows : p_common;
	const int ldb = p_transpose_y ? p_common : p_cols;

	float alpha = 1.f;
	float beta = 0.f;


	hipblasSgemm(handle,
		transpose_x, transpose_y,
		p_rows, p_common, p_cols,
		&alpha, // 1
		p_x, lda,
		p_y, ldb,
		&beta, // 0
		p_z, p_cols);

	hipDeviceSynchronize();
}

void tensor_operator_gpu::const_mul(float* p_x, const float p_y, float* p_z, const int p_size)
{
	call_kernel(const_mul_kernel, p_x, p_y, p_z, p_size);
}

void tensor_operator_gpu::const_div(const float p_x, float* p_y, float* p_z, const int p_size)
{
	call_kernel(const_div_kernel, p_x, p_y, p_z, p_size);
}

void tensor_operator_gpu::call_kernel(void(* kernel)(const float*, const float*, float*, int), float* p_x, float* p_y, float* p_z, const int p_size)
{
	const int blockSize = 256;
	int gridSize = static_cast<int>(ceil(static_cast<float>(p_size) / blockSize));

	kernel<<<gridSize, blockSize >> >(p_x, p_y, p_z, p_size);
	const hipError_t cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cuda_status));
	}
	hipError_t error = hipDeviceSynchronize();
}

void tensor_operator_gpu::call_kernel(void(* kernel)(const float*, const float, float*, const int), float* p_x, const float p_y, float* p_z, const int p_size)
{
	const int blockSize = 256;
	int gridSize = static_cast<int>(ceil(static_cast<float>(p_size) / blockSize));

	kernel <<<gridSize, blockSize >> >(p_x, p_y, p_z, p_size);
	const hipError_t cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cuda_status));
	}
	hipError_t error = hipDeviceSynchronize();
}

void tensor_operator_gpu::call_kernel(void(* kernel)(const float, const float*, float*, const int), const float p_x, float* p_y, float* p_z, const int p_size)
{
	const int blockSize = 256;
	int gridSize = static_cast<int>(ceil(static_cast<float>(p_size) / blockSize));

	kernel <<<gridSize, blockSize >> >(p_x, p_y, p_z, p_size);
	const hipError_t cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cuda_status));
	}
	hipError_t error = hipDeviceSynchronize();
}

tensor_operator_gpu::tensor_operator_gpu()
= default;


tensor_operator_gpu::~tensor_operator_gpu()
= default;

